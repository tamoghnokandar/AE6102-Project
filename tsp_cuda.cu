
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>

using namespace std;

const int THREADS_PER_BLOCK = 1024;
const int BLOCKS = 50;

const int MAXN = 16;
const int INF = 1e9;
const int MIN_EDGE_WEIGHT = 1;
const int MAX_EDGE_WEIGHT = 10;


long long factorial[MAXN+1];

__managed__ int block_optimal_values[BLOCKS];
// __managed__ int block_optimal_paths[BLOCKS][MAXN+1];
__managed__ int block_optimal_permutation[BLOCKS];

/////////////////// Host Functions ///////////////////

__host__ int random(int l, int r) {
  return l + rand()%(r-l+1);
}

__host__ void precompute_factorial() {
	factorial[0] = 1;

	for(int i=1;i<=MAXN;i++)
	{
		factorial[i] = i * factorial[i-1];
	}
}

__host__ void assign_edge_weights(int* matrix, int N) {
	for (int i = 0 ; i < N ; i++) {
		for (int j = i+1 ; j < N ; j++) {
			matrix[i*N + j] = random(MIN_EDGE_WEIGHT,MAX_EDGE_WEIGHT);
			matrix[j*N + i] = matrix[i*N + j];
		}
		matrix[i*N + i] = 0;
	}
}


__host__ void print_matrix(int* matrix, int N) {
	for(int i=0; i<N; i++) {
		for(int j=0; j<N; j++) {
			cout << matrix[i*N + j] << " ";
		}
		printf("\n");
	}
}

/////////////////// Device Functions ///////////////////

__device__ void swap(int &a, int &b) {
	int temp = a;
	a = b;
	b = temp;
}

__host__ __device__ long long fact(int n) {
	long long ans = 1;
	for(int i=1;i<=n;i++) {
		ans *= i;
	}
	return ans;
}

__device__ bool nxt_permutation(int *arr, int n) {
	bool nxt_permutation_possible = false;

	int fi = -1;
	
	for(int i=n-2;i>=0;i--) {
		if(arr[i+1] > arr[i]) {
			nxt_permutation_possible = true;
			fi = i;
			break;
		}
	}

	if(!nxt_permutation_possible)return false;

	int next_greater_ele = arr[fi+1], next_greater_ele_ind = fi+1;

	for(int i=fi+2;i<n;i++) {
		if(arr[i] > arr[fi] && arr[i] < next_greater_ele) {
			next_greater_ele = arr[i];
			next_greater_ele_ind = i;
		}
	}

	swap(arr[fi],arr[next_greater_ele_ind]);

	//Reverse
	int li = fi+1, ri = n-1;
	while(li < ri) {
		swap(arr[li],arr[ri]);
		li++;
		ri--;
	}

	return true;
}


__device__ int find_path_cost(int* matrix, int* arr, int arrsize, int n) {   
	int cost = 0;
	for(int i=1; i<arrsize; i++) {
        int to = arr[i];
        int from = arr[i-1];
		cost += matrix[to*n + from];
	}
	return cost;
}

/////////////////// Global Functions ///////////////////

//Input array should be sorted
__host__ __device__ bool nth_permutation(int *arr, int arrsize, long long n) {
	if(n>fact(arrsize))return false;

    // Assuming arrSize = N+1
	bool taken[MAXN];

	for(int i=0; i<arrsize; i++) taken[i] = false;
	
	int *ans = new int[arrsize];

	for(int i=0; i<arrsize; i++) {
		int cn = 1;
		long long cval = fact(arrsize-1-i);

		while(cval<n) {
			cn++;
			cval=(long long)cn*cval;
			cval=(long long)cval/(cn-1);
		}

		long long pval = cval*(cn-1)/cn;
		n -= pval;

		for(int j=0; j<arrsize; j++) {
			if(!taken[j]) {
				cn--;
				if(cn==0) {
					ans[i] = arr[j];
					taken[j] = true;
					break;
				}
			}
		}
	}

	for(int i=0; i<arrsize; i++) {
		arr[i] = ans[i];
	}
	free(ans);
	return true;
}

__global__ void tsp_cuda(int* matrix, int* path, long long* factorials, int N) {

	__shared__ int thread_optimal_values[THREADS_PER_BLOCK];
    // __shared__ int* thread_optimal_paths[THREADS_PER_BLOCK];
	__shared__ int thread_optimal_permutation[THREADS_PER_BLOCK];

    int thread = threadIdx.x + blockIdx.x * blockDim.x;

    thread_optimal_values[threadIdx.x] = INF;
    // thread_optimal_paths[threadIdx.x] = new int[N+1];

    long long iter_per_thread = factorials[N-1] / (BLOCKS * THREADS_PER_BLOCK);

    int arr[MAXN-1];
    for (int i = 1; i < N; i++) arr[i-1] = path[i];

	long long start_perm = (thread * iter_per_thread) + 1;
	thread_optimal_permutation[threadIdx.x] = start_perm;

    nth_permutation(arr, N-1, start_perm);

	// Last thread of all handles the permutations not entirely divisible by the total threads in all blocks
	if (thread == (BLOCKS * THREADS_PER_BLOCK) - 1) {
		iter_per_thread += factorials[N-1] % (BLOCKS * THREADS_PER_BLOCK);
	}

    long long iter = 0;
    do {
        
        int temp_path[MAXN+1];
        temp_path[0] = 0;
        for (int i = 1; i < N; i++) temp_path[i] = arr[i-1];
        temp_path[N] = 0;


        int val = find_path_cost(matrix, temp_path, N+1, N);

        if(val < thread_optimal_values[threadIdx.x])
		{
			thread_optimal_values[threadIdx.x] = val;
            // for (int i = 0; i < N+1; i++) thread_optimal_paths[threadIdx.x][i] = temp_path[i];
			thread_optimal_permutation[threadIdx.x] = start_perm + iter;
		}

        iter++;
        nxt_permutation(arr, N-1);
        
    } while (iter < iter_per_thread);

    __syncthreads();

	if (threadIdx.x == 0) {
        int optimal_cost = INF;
        for (int i = 0; i < THREADS_PER_BLOCK; i++) {
            if (thread_optimal_values[i] < optimal_cost) {
                optimal_cost = thread_optimal_values[i];
				block_optimal_values[blockIdx.x] = thread_optimal_values[i];
                // for (int j = 0; j < N+1; j++) {
                //     block_optimal_paths[blockIdx.x][j] = thread_optimal_paths[i][j];
                // }
				block_optimal_permutation[blockIdx.x] = thread_optimal_permutation[i];
            }           
        }
    }
}


//////////////////////////////////////////////////////////////

int main(int argc, char **argv) {

    const int N = stoi(argv[1]);

    precompute_factorial();


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int* matrix = new int[N*N];


    int path[N+1];
    path[0] = 0;
    path[N] = 0;
    for (int i = 1; i < N; i++) path[i] = i;

    assign_edge_weights(matrix, N);

    // print_matrix(matrix, N);
	
	for (int i = 0; i < BLOCKS; i++){
		block_optimal_values[i] = INF;
	}

    int *dev_matrix, *dev_path;
    long long *dev_factorial;
    int mat_size = N*N*sizeof(int);
    int path_size = (N+1)*sizeof(int);
    int factorial_size = (MAXN+1)*sizeof(long long);

    hipMalloc((void **)&dev_matrix, mat_size);
    hipMalloc((void **)&dev_path, path_size);
    hipMalloc((void **)&dev_factorial, factorial_size);

    hipEventRecord(start);

    // Copy inputs from host to device
    hipMemcpy(dev_matrix, matrix, mat_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_path, path, path_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_factorial, factorial, factorial_size, hipMemcpyHostToDevice);

    hipDeviceSetLimit(hipLimitMallocHeapSize, 128*1024*1024);

    // Launch the TSP kernel
	tsp_cuda<<<BLOCKS, THREADS_PER_BLOCK>>>(dev_matrix, dev_path, dev_factorial, N);

	hipDeviceSynchronize();
	hipDeviceSynchronize();

	int optimal_cost = INF;
	long long optimal_permutation;
	for (int i = 0; i < BLOCKS; i++) {
		if (block_optimal_values[i] < optimal_cost) {
			optimal_cost = block_optimal_values[i];
			// for (int j = 0; j < N+1; j++) {
			// 	path[j] = block_optimal_paths[i][j];
			// }
			optimal_permutation = block_optimal_permutation[i];
		}	
	}

	int arr[MAXN-1];
    for (int i = 1; i < N; i++) arr[i-1] = path[i];
	nth_permutation(arr, N-1, optimal_permutation);

	for (int i = 1; i < N; i++) path[i] = arr[i-1];

    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

	printf("%f\n", milliseconds*0.001);

    // printing the minimum cost path
    printf("Minimum Cost Path: ");
    for (int i = 0; i < N+1; i++) {
        printf("%d ", path[i]);
    }
    printf("\n");

    // printing the minimum cost path
    int cost = 0;
    for(int i=1; i<N+1; i++) {
        cost += matrix[path[i]*N + path[i-1]];
    }
    printf("Path cost: %d \n", cost);

    // printing the run-time
    // printf("Time taken: %f s\n", milliseconds*0.001);

    hipFree(dev_matrix);
    hipFree(dev_path);
	hipFree(dev_factorial);
}
